#include <hip/hip_runtime.h>
#include <isostream>

#define CK(call) do {
    hipError_t _e = (call);
    if (_e != hipSuccess {
        std::cerr << "CUDA error: " << hipGetErrorString(_e) << "at" << __FILE__ << ";" << __LINE__ << std::endl;
        std::exit(1);
    }
} while(0)


int main() {

}