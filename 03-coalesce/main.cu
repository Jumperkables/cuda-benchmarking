#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>


#define CK(call) do { \
  hipError_t _e = (call); \
  if (_e != hipSuccess) { \
    std::cerr << "CUDA error: " << hipGetErrorString(_e) \
              << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
    std::exit(1); \
  } \
} while(0)


__global__ void clear_vector_coalesced(float* v, int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n){
        v[id] = 0.0;    // Coalesced because when a thread comes to access this, it is naturally laid out such that each thread will be taking from contiguous memory
    }
}

__global__ void clear_vector_non_coalesced(float* v, int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    id *= 4;
    if (id < n){
        v[id] = 0.0;
        v[id + 1] = 0.0;
        v[id + 2] = 0.0;
        v[id + 3] = 0.0;
    }
}


int main() {
    // Instantiate vector
    int N = 1<<30;
    float *v = new float[N];
    float *w = new float[N];

    // Instantiate CUDA block and grid size
    int blockSize = 128;   // Multiple of 32
    int gridSize = (N + blockSize - 1) / blockSize;

    // Time the coalesced /////////////////////////////////////////////
    // Allocate and copy device buffers for the device code objects
    size_t bytes = N * sizeof(float);
    float *d_v = nullptr;
    CK(hipMalloc(&d_v, bytes));
    CK(hipMemcpy(d_v, v, bytes, hipMemcpyHostToDevice));

    // Set up timer and run code
    hipEvent_t start1, stop1;
    CK(hipEventCreate(&start1));
    CK(hipEventCreate(&stop1));
    CK(hipEventRecord(start1));
    clear_vector_coalesced<<<gridSize, blockSize>>>(d_v, N);
    CK(hipGetLastError());
    CK(hipEventRecord(stop1));
    CK(hipEventSynchronize(stop1));

    // output elapse time
    float ms1 = 0.0f;
    CK(hipEventElapsedTime(&ms1, start1, stop1));
    std::cout << "Coalesced time: " << ms1 << "ms\n";
    CK(hipEventDestroy(start1));
    CK(hipEventDestroy(stop1));

    // CUDA Cleanup
    CK(hipFree(d_v));
    // clean host array
    delete [] v;

    // Time the uncoalesced /////////////////////////////////////////////
    // Allocate and copy device buffers for the device code objects
    float *d_w = nullptr;
    CK(hipMalloc(&d_w, bytes));
    CK(hipMemcpy(d_w, w, bytes, hipMemcpyHostToDevice));

    // Set up timer and run code
    hipEvent_t start2, stop2;
    CK(hipEventCreate(&start2));
    CK(hipEventCreate(&stop2));
    CK(hipEventRecord(start2));
    clear_vector_non_coalesced<<<gridSize, blockSize>>>(d_w, N);
    CK(hipGetLastError());
    CK(hipEventRecord(stop2));
    CK(hipEventSynchronize(stop2));

    // output elapse time
    float ms2 = 0.0f;
    CK(hipEventElapsedTime(&ms2, start2, stop2));
    std::cout << "Coalesced time: " << ms2 << "ms\n";
    CK(hipEventDestroy(start2));
    CK(hipEventDestroy(stop2));

    // CUDA Cleanup
    CK(hipFree(d_w));
    // clean host array
    delete [] w;
}