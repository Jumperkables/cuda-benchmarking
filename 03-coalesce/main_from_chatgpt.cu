#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdint>

#define CK(call) do { \
  hipError_t _e = (call); \
  if (_e != hipSuccess) { \
    std::cerr << "CUDA error: " << hipGetErrorString(_e) \
              << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
    std::exit(1); \
  } \
} while(0)

// ===== kernels you said you'd use =====
__global__ void clear_coalesced_gridstride(float* __restrict__ v, size_t n){
    size_t i = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = (size_t)blockDim.x * gridDim.x;
    for (; i < n; i += step) v[i] = 0.0f;
}

__global__ void clear_strided_gridstride(float* __restrict__ v, size_t n, int s){
    size_t i = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = (size_t)blockDim.x * gridDim.x;
    for (; i < n; i += step) {
        size_t j = i * (size_t)s;     // scatter by stride s
        if (j < n) v[j] = 0.0f;       // write only if in range
    }
}

// small helper
static inline double gbps(size_t bytes, float ms){
    // bytes / (ms/1e3) / 1e9
    return (double)bytes / (double)ms * 1e-6;
}

int main(int argc, char** argv){
    // Params (override via CLI: N stride)
    size_t N = (argc > 1) ? std::strtoull(argv[1], nullptr, 10) : (1ull << 28); // 268,435,456 elements (~1.0 GiB)
    int stride = (argc > 2) ? std::atoi(argv[2]) : 32;                          // try 1 (coalesced) vs 32/64 (non-coalesced)

    std::cout << "N = " << N << " elements (" << (N*sizeof(float))/ (1024.0*1024.0*1024.0)
              << " GiB), stride = " << stride << "\n";

    // Device alloc
    float* d_buf = nullptr;
    size_t bytes = N * sizeof(float);
    CK(hipMalloc(&d_buf, bytes));

    // Launch config (same for both kernels)
    int blockSize = 256;
    // cap grid to something reasonable (max 65535 for legacy 1D grid; you can pick larger with modern launches, but this is fine)
    int gridSize = (int)std::min(
        (size_t)65535,
        (N + (size_t)blockSize - 1) / (size_t)blockSize
    );

    // Warm-up (pay JIT & clock ramp)
    clear_coalesced_gridstride<<<gridSize, blockSize>>>(d_buf, N);
    CK(hipGetLastError());
    CK(hipDeviceSynchronize());
    clear_strided_gridstride<<<gridSize, blockSize>>>(d_buf, N, stride);
    CK(hipGetLastError());
    CK(hipDeviceSynchronize());

    // --- Time coalesced ---
    hipEvent_t s1, e1;
    CK(hipEventCreate(&s1)); CK(hipEventCreate(&e1));
    CK(hipEventRecord(s1));
    clear_coalesced_gridstride<<<gridSize, blockSize>>>(d_buf, N);
    CK(hipGetLastError());
    CK(hipEventRecord(e1));
    CK(hipEventSynchronize(e1));
    float ms1 = 0.0f;
    CK(hipEventElapsedTime(&ms1, s1, e1));
    CK(hipEventDestroy(s1)); CK(hipEventDestroy(e1));

    // Effective bytes written: N floats
    double gbps1 = gbps(bytes, ms1);
    std::cout << "Coalesced: " << ms1 << " ms, "
              << gbps1 << " GB/s\n";

    // --- Time strided ---
    // Note: this kernel writes about ceil(N/stride) floats
    size_t written_elems_strided = (N + (size_t)stride - 1) / (size_t)stride;
    size_t bytes_strided = written_elems_strided * sizeof(float);

    hipEvent_t s2, e2;
    CK(hipEventCreate(&s2)); CK(hipEventCreate(&e2));
    CK(hipEventRecord(s2));
    clear_strided_gridstride<<<gridSize, blockSize>>>(d_buf, N, stride);
    CK(hipGetLastError());
    CK(hipEventRecord(e2));
    CK(hipEventSynchronize(e2));
    float ms2 = 0.0f;
    CK(hipEventElapsedTime(&ms2, s2, e2));
    CK(hipEventDestroy(s2)); CK(hipEventDestroy(e2));

    double gbps2 = gbps(bytes_strided, ms2);
    std::cout << "Strided (s=" << stride << "): " << ms2 << " ms, "
              << gbps2 << " GB/s ("
              << written_elems_strided << " elems written)\n";

    // Cleanup
    CK(hipFree(d_buf));
    return 0;
}